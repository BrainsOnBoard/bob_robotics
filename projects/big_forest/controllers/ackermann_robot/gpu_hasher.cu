#include "hip/hip_runtime.h"
// cuda implementation of hash matching and sequence
#include "navigation/image_database.h"
#include "imgproc/dct_hash.h"
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>

// Standard C includes
#include <ctime>
#include <chrono>
#include <string.h>
#include <bitset>

#include <thrust/device_ptr.h>
#include <thrust/extrema.h>


const int BLOCKSIZE = 256;
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}




// number of blocks = number of rows * 2 - 1, number of threads/block = number of rows (<= blockSize)
__global__ void kernel_order_dist_matrix(int *d_dist_mat, int *d_ordered_dist_mat, int row_n, int col_n) {
    int uid = blockIdx.x*blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int col, row, next_row, next_col;

    if (tid == 0) {
        d_ordered_dist_mat[0] = d_dist_mat[0];
    }
    // each block processes a diagonal line [-1,+1] direction
    // start of teh row - increasing diagonal elements
    if (bid < row_n && tid > 0 && bid > 0) {

        next_row = blockIdx.x - tid;
        next_col = tid;
        // check bounds
        if ((next_row > 0 && next_row < row_n) && (next_col > 0 && next_col < col_n)) {
            d_ordered_dist_mat[uid] = d_dist_mat[next_row*row_n + next_col];
        }

    } else {
        // diagonal line elements decreasing
        next_row = row_n - 1; // last row
        next_col = (blockIdx.x - row_n-1) + tid;       //
        // check bounds
        if ((next_row > 0 && next_row < row_n) && (next_col > 0 && next_col < col_n)) {
            d_ordered_dist_mat[uid] = d_dist_mat[next_row*row_n + next_col];
        }
    }
    __syncthreads();

}

__global__ void kernel_calculate_accumulated_cost_matrix() {

}


__device__ void warp_reduce(volatile int *s_data, int tid) {
    if (tid < 32) {
        s_data[tid] = min(s_data[tid],s_data[tid + 32]);
        s_data[tid] = min(s_data[tid],s_data[tid + 16]);
        s_data[tid] = min(s_data[tid],s_data[tid +  8]);
        s_data[tid] = min(s_data[tid],s_data[tid +  4]);
        s_data[tid] = min(s_data[tid],s_data[tid +  2]);
        s_data[tid] = min(s_data[tid],s_data[tid +  1]);
    }
}




// number of blocks = number of unique elements, number of threads/block = number of rotations (<= blockSize)
__global__ void kernel_calculateMatrixBlock_row(
                                    unsigned long long int hash,          // sequence of hash [should be the block size]
                                    unsigned long long int *d_training_matrix,      // training matrix with [blocksize] rotations for each element
                                    int *d_rotation_dist_matrix_col,                    // output matrix [reduced to best rotations]
                                    int N // number of elements
                                    ) {

    int uid = blockIdx.x*blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    __shared__ unsigned long long int s_rotations[BLOCKSIZE];
    __shared__ int s_dist_mat_row[BLOCKSIZE];
    __shared__ int s_dist_mat_col[BLOCKSIZE];


    s_rotations[tid] = d_training_matrix[uid]; // load each rotation row to each block of share memory
    __syncthreads();
    s_dist_mat_row[tid] = __popc(hash ^ s_rotations[tid]);
    __syncthreads();

    for(unsigned int stride = (blockDim.x/2); stride > 32 ; stride /=2){
        __syncthreads();

        if(tid < stride)
        {
            s_dist_mat_row[tid]  = min(s_dist_mat_row[tid],s_dist_mat_row[tid + stride]);

        }
    }

    warp_reduce(s_dist_mat_row, tid);
    __syncthreads();
    // save the best rotation in the column
    if (tid == 0) {
        d_rotation_dist_matrix_col[bid] = s_dist_mat_row[0];

    }

}


__global__ void kernel_construct_distance_matrix(unsigned long long int *d_sequence, unsigned long long int *d_training_matrix, int *d_distance_matrix, int sequence_size, int N) {
    int uid = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned long long int hash = d_sequence[uid];
    if (uid < sequence_size) {
        kernel_calculateMatrixBlock_row<<<N,BLOCKSIZE>>>(hash, d_training_matrix, &d_distance_matrix[uid*N], N);
    }
    hipDeviceSynchronize();
}

__global__ void kernel_shift_elements(unsigned long long int *d_sequence, unsigned long long int *d_tmp_seq, int d_sequence_size) {
    int uid = blockIdx.x*blockDim.x + threadIdx.x;

    if (uid < d_sequence_size-1 ) {
        d_tmp_seq[uid] = d_sequence[uid+1];
        __syncthreads();
        d_sequence[uid] = d_tmp_seq[uid];
    }



}


class GPUHasher
{
    public:
    GPUHasher() {}

    // allocate memory using the training matrix [num_elements x num_rotations]
    void initGPU(unsigned long long int *l_hash_mat, int hash_mat_size, int sequence_size, int num_rotations) {

        N = hash_mat_size;
        num_rows = N/num_rotations;
        num_cols = num_rotations;
        d_sequence_size = sequence_size;


        l_cost_matrix = (int *) malloc(N * sizeof(int));
        gpuErrchk( hipMalloc(&d_tmp_seq, d_sequence_size*sizeof(unsigned long long int)) );
        gpuErrchk( hipMalloc(&d_tmp_cost_mat, d_sequence_size*N*sizeof(int))); // this will store all the sequence matrices
        gpuErrchk( hipMalloc(&d_cost_matrix, d_sequence_size*num_rows*sizeof(int)));
        gpuErrchk( hipMalloc(&d_sequence, d_sequence_size*sizeof(unsigned long long int)));
        gpuErrchk( hipMalloc(&d_hash_mat, N*sizeof(unsigned long long int)));
        gpuErrchk( hipMemcpy(d_hash_mat, l_hash_mat, N*sizeof(unsigned long long int), hipMemcpyHostToDevice));

    }

    // upload a hash sequence to the gpu
    void uploadSequence(unsigned long long int *sequence) {
        l_sequence = sequence;
        gpuErrchk( hipMemcpy(d_sequence, sequence, d_sequence_size*sizeof(unsigned long long int), hipMemcpyHostToDevice));
    }

    // append one element to the hash sequence and pop front
    void addToSequence(unsigned long long int *hash) {

        kernel_shift_elements<<<(d_sequence_size+255/256),BLOCKSIZE>>>(d_sequence,d_tmp_seq, d_sequence_size);
        hipMemcpy(&d_sequence[BLOCKSIZE-1], hash, sizeof(unsigned long long int), hipMemcpyHostToDevice);
    }

    // get distance matrix between two datasets
    void getDistanceMatrix(unsigned long long int *sequence) {
        uploadSequence(sequence);

        kernel_construct_distance_matrix<<<(d_sequence_size+255/256), d_sequence_size>>>(d_sequence, d_hash_mat, d_cost_matrix, d_sequence_size, num_rows);
    }

    // gets the current distance matrix
    void getDistanceMatrix() {
        //kernel_sequence_XOR<<<(N+255/256),BLOCKSIZE>>>(d_sequence, d_hash_mat, d_tmp_cost_mat, N, d_sequence_size);
        //kernel_buildCostMatrix<<<(d_sequence_size+255/256),BLOCKSIZE>>>(d_tmp_cost_mat, d_cost_matrix, N, num_rows, d_sequence_size, num_cols);
        kernel_construct_distance_matrix<<<(d_sequence_size+255/256),d_sequence_size>>>(d_sequence, d_hash_mat, d_cost_matrix, d_sequence_size, num_rows);

        hipDeviceSynchronize();

    }

    cv::Mat downloadDistanceMatrix() {
        gpuErrchk( hipMemcpy(l_cost_matrix, d_cost_matrix, num_rows*d_sequence_size*sizeof(int), hipMemcpyDeviceToHost) );
        cv::cuda::GpuMat gpu_mat({d_sequence_size, num_rows, CV_32SC1, d_cost_matrix});
        cv::Mat host_mat;
        gpu_mat.download(host_mat);
        host_mat.convertTo(host_mat,CV_8UC1);

        return host_mat;
    }

    ~GPUHasher() {
        hipFree(d_hash_mat);
        hipFree(d_sequence);
        hipFree(d_tmp_cost_mat);
        hipFree(d_cost_matrix);
    }

    private:

    const int BLOCKSIZE = 256;
    unsigned long long int *l_hash_mat; // host hash mat
    unsigned long long int *d_hash_mat; // device hash mat
    unsigned long long int *l_sequence; // sequence on host
    unsigned long long int *d_sequence; // sequence on device
    unsigned long long int *d_tmp_seq;  // temporary sequence for shift elements
    int *d_tmp_cost_mat;                // sequence of cost matrices with rotations
    int *l_cost_matrix;                 // final cost matrix on host
    int *d_cost_matrix;                 // final cost matrix on device
    unsigned int d_sequence_size;       // size of the sequence
    int N;                              // total size of a hash matrix with rotations
    int num_rows;                       // number of unique elements in a hash matrix
    int num_cols;                       // number of rotations in the training matrix

};

