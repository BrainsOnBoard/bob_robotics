#include "hip/hip_runtime.h"
#pragma once
// cuda implementation of hash matching and sequence
#include "gpu_image_matching_kernels.cuh"
#include "navigation/image_database.h"
#include <opencv2/core.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/imgcodecs.hpp>
#include "imgproc/roll.h"
#include "imgproc/gpu_dct.h"
// Standard C includes
#include <ctime>
#include <chrono>
#include <string.h>
#include <bitset>
#include <bit>
#include <cstdint>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>
#include <hipblas.h>


class GPUHasher
{
    public:
    GPUHasher() {}

    // allocate memory using the training matrix [num_elements x num_rotations]
    void initGPU(unsigned long long int *l_hash_mat,
                int hash_mat_size,
                int sequence_size,
                int num_rotations,
                int img_width,
                int img_height
                ) {

        N = hash_mat_size;
        num_rows = N/num_rotations;
        num_cols = num_rotations;
        d_sequence_size = sequence_size;
        l_cost_matrix = (int *) malloc(N * sizeof(int));
        l_best_row = (unsigned long long int *) malloc(num_rotations * sizeof(unsigned long long int));
        l_sequence = (unsigned long long int *) malloc(d_sequence_size * sizeof(unsigned long long int));
        l_rot_img_data = (uchar *) malloc(img_height*img_width * sizeof(uchar));
        l_accumulated_cost_matrix = (int *) malloc(d_sequence_size * num_rows * sizeof(int));
        m_image_width = img_width;
        m_image_height = img_height;


        gpuErrchk( hipMalloc(&d_image, img_height*img_width*sizeof(uchar)));
        gpuErrchk( hipMalloc(&d_rolled_image, img_height*img_width*sizeof(uchar)));
        gpuErrchk( hipMalloc(&d_ordered_cost_mat, d_sequence_size*num_rows*sizeof(int)));
        gpuErrchk( hipMalloc(&d_tmp_seq, d_sequence_size*sizeof(unsigned long long int)) );

        // for rot dist mat
        gpuErrchk( hipMalloc(&training_route, N/num_rotations*sizeof(unsigned long long int)) );
        gpuErrchk( hipMalloc(&current_rotations, num_rotations*sizeof(unsigned long long int)) );
        gpuErrchk( hipMalloc(&rot_dist_mat, num_rotations*(N/num_rotations)*sizeof(int)) );


        gpuErrchk( hipMalloc(&d_cost_matrix, d_sequence_size*num_rows*sizeof(int)));
        gpuErrchk( hipMalloc(&d_index_matrix, d_sequence_size*num_rows*sizeof(int)));
        gpuErrchk( hipMalloc(&d_index_matrix_ord, d_sequence_size*num_rows*sizeof(int)));
        gpuErrchk( hipMalloc(&d_accumulated_cost_mat, d_sequence_size*num_rows*sizeof(int)));
        gpuErrchk( hipMalloc(&d_accumulated_cost_mat_ord, d_sequence_size*num_rows*sizeof(int)));
        gpuErrchk( hipMalloc(&d_sequence, d_sequence_size*sizeof(unsigned long long int)));
        gpuErrchk( hipMalloc(&d_hash_mat, N*sizeof(unsigned long long int)));
        gpuErrchk( hipMemcpy(d_hash_mat, l_hash_mat, N*sizeof(unsigned long long int), hipMemcpyHostToDevice));
        gpuErrchk( hipMalloc(&d_rolled_images, num_rotations*img_height*img_width*sizeof(float)));

        // initialize GPU sequence matcher
        kernel_fill_index_matrix<<<num_rows, d_sequence_size>>>(d_index_matrix_ord);
        hipDeviceSynchronize();
        kernel_order_dist_matrix<<<num_rows, d_sequence_size, BLOCKSIZE*2>>>(d_index_matrix_ord, d_index_matrix, d_sequence_size, num_rows);
        hipDeviceSynchronize();


        kernel_get_distmat_column<<<((N/num_rotations)+255)/256,256>>>(d_hash_mat, training_route);

        //getDCTMatrix(img_height, img_width);
        std::cout << " GPU initialized" << std::endl;
    }

    void init_GPU_for_single_match(std::vector<cv::Mat> &training_imgs,
                                  std::vector<cv::Mat> &test_images,
                                  int N_training,
                                  int N_testing,
                                  cv::Size size,
                                  bool isSequence,
                                  int sequence_size = 128,
                                  bool hash_only = true) {
        d_sequence_size = sequence_size;
        unsigned long long int l_training_hashes[N_training];
        unsigned long long int l_testing_hashes[N_testing];
        img_rows = size.height;
        img_cols = size.width;
        this->N_training = N_training;
        this->N_testing = N_testing;
        num_rows = N_training;
        GpuDct gdct(size.width);

        gpuErrchk( hipMalloc(&d_training_hashes, N_training*sizeof(unsigned long long int)));
        gpuErrchk( hipMalloc(&d_testing_hashes, N_testing*sizeof(unsigned long long int)));
        gpuErrchk( hipMalloc(&d_single_distance_matrix, N_testing*N_training*sizeof(int)));
        ull_pointers.push_back(d_training_hashes);
        ull_pointers.push_back(d_testing_hashes);
        i_pointers.push_back(d_single_distance_matrix);

        if (!hash_only) {
            gpuErrchk( hipMalloc(&d_training_images, N_training*size.height*size.width*sizeof(float)));
            gpuErrchk( hipMalloc(&d_testing_images, N_testing*size.height*size.width*sizeof(float)));
            gpuErrchk( hipMalloc(&d_temp, size.height*size.width*N_training*sizeof(float)));
            gpuErrchk( hipMalloc(&d_reduced_blocks, N_training*size.height*sizeof(float)));
            gpuErrchk( hipMalloc(&d_dist_mat_PM, N_training*N_testing*sizeof(float)));

            for (int i = 0; i < training_imgs.size(); i++) {
                cv::Mat curr_img = training_imgs[i];
                gpuErrchk(hipMemcpy(d_training_images+(i*size.width*size.height),
                        reinterpret_cast<float*>(curr_img.data),
                        (size.height*size.width)*sizeof(float),
                        hipMemcpyHostToDevice));

            }

            for (int i = 0; i < test_images.size(); i++) {
                cv::Mat curr_img = test_images[i];
                gpuErrchk(hipMemcpy(d_testing_images+i*size.height*size.width,
                        reinterpret_cast<float*>(curr_img.data),
                        (size.height*size.width)*sizeof(float),
                        hipMemcpyHostToDevice));

            }
            f_pointers.push_back(d_training_images);
            f_pointers.push_back(d_testing_images);
            f_pointers.push_back(d_temp);
            f_pointers.push_back(d_reduced_blocks);
            f_pointers.push_back(d_dist_mat_PM);
        }

        upload_hash_database(training_imgs, d_training_hashes);
        upload_hash_database(test_images, d_testing_hashes);


        std::cout << " gpu for singles matches initialized " << N_testing << std::endl;
    }

     // upload a vector of images to the GPU
     static void upload_hash_database(std::vector<cv::Mat> images, unsigned long long int *hashes_d_ptr) {
        GpuDct gpu_dct(images[0].size().width);
        std::vector<std::bitset<64>> hashes = gpu_dct.batch_dct(images);
        hipDeviceSynchronize();
        upload_hash_database(hashes, hashes_d_ptr);
    }

    // upload hashes to the gpu
    static void upload_hash_database(std::vector<std::bitset<64>> hashes, unsigned long long int *hashes_d_ptr) {
        int num_hash = hashes.size();
        unsigned long long int ull_hashes[num_hash];
        hipMalloc(&hashes_d_ptr, num_hash*sizeof(unsigned long long int));
        for (int i = 0; i < num_hash; i++) {
            std::cout << hashes[i] << std::endl;
            ull_hashes[i] = hashes[i].to_ullong();
        }
        hipMemcpy(hashes_d_ptr, ull_hashes, num_hash*sizeof(unsigned long long int), hipMemcpyHostToDevice);
    }


    void upload_hash_rotations_gpu(std::vector<std::bitset<64>> rots, int totalRotations, unsigned long long int *d_rotations) {

        unsigned long long int rotations[totalRotations];
        for (int i = 0; i < rots.size(); i++) {
            rotations[i] = rots[i].to_ullong();
        }

        gpuErrchk( hipMemcpy(d_rotations, rotations, totalRotations*sizeof(unsigned long long int), hipMemcpyHostToDevice));

    }

    cv::Mat calculate_rotation_dist_matrix(std::vector<std::bitset<64>> hash_rots, int totalRotations) {

        int N_sample = N/totalRotations;
        upload_hash_rotations_gpu(hash_rots, totalRotations, current_rotations); // get rotation hashes

        kernel_get_distmat_from_rotations<<<(N_sample+255)/256,256>>>(current_rotations, training_route, rot_dist_mat);

        //int l_rot_dist_mat[N_sample*totalRotations];
        //gpuErrchk( hipMemcpy(l_rot_dist_mat, rot_dist_mat, N_sample*totalRotations*sizeof(int), hipMemcpyDeviceToHost) );
        cv::cuda::GpuMat gpu_mat({ N_sample, totalRotations,CV_32SC1, rot_dist_mat});
        cv::Mat host_mat;
        gpu_mat.download(host_mat);
        return host_mat;
    }

    void upload_database(std::vector<cv::Mat> images, int w, int h) {

        pm_width = w;
        pm_height = h;
        N_images = images.size();
        int num_rotations = w; // rotations in the image = width of image
        num_blocks = (h*w+BLOCKSIZE-1)/BLOCKSIZE;
        hipMalloc(&d_temp, h*w*num_rotations*sizeof(float));
        hipMalloc(&d_reduced_blocks, num_rotations*num_blocks*sizeof(float));
        hipMalloc(&d_SSD_rotations, num_rotations*sizeof(float));
        hipMalloc(&d_dist_mat_PM, N_images*num_rotations*sizeof(float));
        hipMalloc(&d_images, (h*w)*N_images*sizeof(float));

        for (int i = 0; i < images.size(); i++) {
            cv::Mat curr_img = images[i];
            cv::cvtColor(curr_img, curr_img, cv::COLOR_BGR2GRAY);
            cv::resize(curr_img, curr_img, {w, h},2);
            curr_img.convertTo(curr_img, CV_32FC1,(1.0)/255.0);
            hipMemcpy(d_images+i*h*w,
                       reinterpret_cast<float*>(curr_img.data),
                       (h*w)*sizeof(float),
                       hipMemcpyHostToDevice);
        }
    }



    // print a matrix on the GPU
    template<typename T>
    void printMatrix(T *d_matrix,int rows, int cols, int rowLim, int colLim) {
        T tmp[rows*cols];
        hipMemcpy(tmp, d_matrix, rows*cols*sizeof(T), hipMemcpyDeviceToHost);
        std::cout << std::fixed;
        std::cout << std::setprecision(4);
        std::cout << "-------------------------------------" << std::endl;
        std::cout << "-------------Matrix start------------" << std::endl;
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                if (i < rowLim && j < colLim) {
                    std::cout << "["<< tmp[i*cols+j] << "]";
                } else {
                    break;
                }

            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
        std::cout << "-------------Matrix end--------------" << std::endl;
        std::cout << "-------------------------------------" << std::endl;
    }

    // rolls images to get all the rotations - return rolled images on host
    static void get_rotations(const cv::Mat &image, float *d_rotated, const int num_rotations) {

        int cols = image.cols;
        int rows = image.rows;
        auto type = image.type();
        float *d_img;
        cv::Mat curr_img;
        image.convertTo(curr_img, CV_32FC1,(1.0)/255.0);
        gpuErrchk( hipMalloc(&d_img, rows*cols*sizeof(float)) );
        gpuErrchk( hipMemcpy(d_img, reinterpret_cast<float*>(curr_img.data), rows*cols*sizeof(float), hipMemcpyHostToDevice) );

        std::vector<hipStream_t> cuda_streams;
        for (int i = 0; i < num_rotations; i++) {
            hipStream_t stream;
            hipStreamCreate( &stream);
            cuda_streams.push_back(stream);
            int offset = i*cols*rows;
            kernel_roll_image<<<rows, cols, cols*sizeof(float), cuda_streams[i] >>>(d_rotated+offset, d_img, cols,i);
        }
        for (int i = 0; i < num_rotations; i++) {
            hipStreamDestroy(cuda_streams[i]);
        }
        hipDeviceSynchronize();
        hipFree(d_img);
    }


    void get_rotations(const cv::Mat &image, const int num_rotations) {

        int cols = image.cols;
        int rows = image.rows;
        auto type = image.type();
        float *d_img;
        cv::Mat curr_img;
        image.convertTo(curr_img, CV_32FC1,(1.0)/255.0);
        gpuErrchk( hipMalloc(&d_img, rows*cols*sizeof(float)) );
        gpuErrchk( hipMemcpy(d_img, reinterpret_cast<float*>(curr_img.data), rows*cols*sizeof(float), hipMemcpyHostToDevice) );

        std::vector<hipStream_t> cuda_streams;
        for (int i = 0; i < num_rotations; i++) {
            hipStream_t stream;
            hipStreamCreate( &stream);
            cuda_streams.push_back(stream);
        }

        for (int i = 0; i < num_rotations; i++) {
            int offset = i*cols*rows;
            kernel_roll_image<<<rows, cols, cols*sizeof(float), cuda_streams[i] >>>(d_rolled_images+offset, d_img, cols,i);
        }
        hipDeviceSynchronize();
        hipFree(d_img);
    }



    void get_hash_rotation_matrix(const std::vector<cv::Mat> &images, const int num_rotations) {

        int cols = num_rotations;
        int rows = num_rotations;
        int n = num_rotations;
        int im_size = n*n;
        GpuDct gct(num_rotations);
        int n_images = images.size();

        std::vector<std::vector<std::bitset<64>>> hash_matrix;
        float *d_img_array; hipMalloc(&d_img_array, n_images * n * n*sizeof(float));

        for (int i = 0; i < n_images; i++) {
            cv::Mat curr_img;
            cv::Mat image = images[i];
            cv::resize(image, curr_img, {n,n});
            curr_img.convertTo(curr_img, CV_32FC1,(1.0)/255.0);
            gpuErrchk( hipMemcpy(d_img_array + i * im_size, reinterpret_cast<float*>(curr_img.data), n*n*sizeof(float), hipMemcpyHostToDevice) );
        }

        std::vector<hipStream_t> streams;
        for (int i = 0; i < num_rotations; i++) {
            hipStream_t stream;
            hipStreamCreate(&stream);
            streams.push_back(stream);
            float *d_rotated_image_array; hipMallocAsync(&d_rotated_image_array, num_rotations*n*n*sizeof(float), streams[i]);
            kernel_roll_image<<<n*n_images, n, n*sizeof(float), streams[i] >>>(d_rotated_image_array, d_img_array, n,i);

            hipStreamSynchronize(streams[i]);
            std::vector<std::bitset<64>> hashes =  gct.gpu_stream_dct( d_rotated_image_array,  n_images, n, streams[i]);
            hipStreamSynchronize(streams[i]);
            hash_matrix.push_back(hashes);
            hipFree(d_rotated_image_array);
            hipStreamDestroy(streams[i]);
        }

        hipDeviceSynchronize();
        std::cout << " done " << hash_matrix[0][0] << std::endl;


    }

     std::vector<std::bitset<64>> get_rotation_hashes(const cv::Mat &image, std::vector<cv::Mat> &rotated_images, const int num_rotations) {

        int n = num_rotations;
        int im_size = n*n;
        cv::Mat resized;
        float *d_img;
        // if image is not square, make it square
        cv::resize(image,resized, {num_rotations,num_rotations});
        resized.convertTo(resized, CV_32FC1,(1.0)/255.0);
        float *d_rotated_images;
        gpuErrchk( hipMalloc(&d_img, im_size*sizeof(float)) );
        gpuErrchk( hipMalloc(&d_rotated_images, im_size*num_rotations*sizeof(float)));

        hipMemcpy(d_img, reinterpret_cast<float*>(resized.data), im_size*sizeof(float), hipMemcpyHostToDevice);
        GpuDct gdct(num_rotations);
        std::vector<std::bitset<64>> rotated_hash_vector;

        for (int i = 0; i < num_rotations; i++) {
            int offset = i*im_size;
            kernel_roll_image<<<n, n, n*sizeof(float) >>>(d_rotated_images+offset, d_img, n,i);
            hipDeviceSynchronize();
            auto hash = gdct.dct(d_rotated_images+offset);
            rotated_hash_vector.push_back(hash);
            cv::Mat host_image(n,n, CV_32FC1);
            hipMemcpy(host_image.data, reinterpret_cast<uchar*>(d_rotated_images+offset), im_size*sizeof(float), hipMemcpyDeviceToHost);
            rotated_images.push_back(host_image);


        }
        hipDeviceSynchronize();

        hipFree(d_rotated_images);

        return rotated_hash_vector;
    }



    int * get_single_hash_difference_matrix(std::vector<std::pair<int,int>> &scores, cv::Mat &distance_matrix) {
        int threads = 256;
        int blocks = int(N_training+threads-1/threads);
        kernel_simple_dist_mat<<<blocks,threads>>>(d_training_hashes, d_testing_hashes, d_single_distance_matrix, N_training, N_testing);


        cv::cuda::GpuMat gpu_mat({N_training, N_testing, CV_32SC1, d_single_distance_matrix});
        cv::Mat temp_mat;
        gpu_mat.download(temp_mat);

        cv::normalize(temp_mat, temp_mat, 0, 255, cv::NORM_MINMAX);
        temp_mat.convertTo(temp_mat, CV_8UC1);
        cv::applyColorMap(temp_mat, temp_mat, cv::COLORMAP_JET);

        scores = get_closest_matches_from_dist_mat( d_single_distance_matrix);
        distance_matrix = temp_mat;
        return d_single_distance_matrix;
    }

    template <typename T>
    std::vector<std::pair<int,int>> get_closest_matches_from_dist_mat(T *dist_mat) {
        std::vector<std::pair<int,int>> scores;
        for (int i = 0; i < N_testing; i++) {
            std::pair<int, int> pair;

            thrust::device_ptr<T> g_ptr =  thrust::device_pointer_cast(&dist_mat[i*N_training]);
            int result_offset = thrust::min_element( g_ptr, g_ptr + (N_training) ) -g_ptr;
            int min_score = *(g_ptr + result_offset);
            pair.first = i;
            pair.second = result_offset;
            scores.push_back(pair);
        }
        return scores;

    }

    cv::Mat get_best_PM_single_match(std::vector<std::pair<int,int>> &scores) {

        int N_blocks = N_training * img_rows;
        //std::cout << "N train = " << N_training << " img rows = " << img_rows << " N blocks = "<< N_blocks << std::endl;
        for (int i = 0; i < N_testing; i++) {
           // if ((i % 100) == 0) { std::cout << "i= " << i << std::endl;}
            // absolute differencing which gives a difference matrix for all images
            kernel_SAD_single_matches<<<N_blocks,BLOCKSIZE>>>
                                            (d_training_images,
                                            d_testing_images+i*img_rows*img_cols,
                                            d_temp,
                                            img_rows,
                                            img_cols,
                                            N_training);

            hipDeviceSynchronize();
            // summing up the values in the difference matrix
            for (int j = 0; j < N_training; j++) {
                kernel_reduce<<<img_rows,BLOCKSIZE>>>(d_temp+j*img_rows*img_cols, d_reduced_blocks+j*img_rows, img_rows*img_cols);
            }

            hipDeviceSynchronize();
            kernel_reduce<<<N_training,img_rows>>>(d_reduced_blocks, d_dist_mat_PM+i*N_training, N_training*img_rows);
            hipDeviceSynchronize();
        }
        hipDeviceSynchronize();

        float* temp_array = (float *) malloc(N_training*N_testing * sizeof(float));
        hipMemcpy(temp_array, d_dist_mat_PM, N_training*N_testing*sizeof(float), hipMemcpyDeviceToHost);
        cv::Mat temp_mat(cv::Size(N_training,N_testing), CV_32FC1, temp_array);
        cv::normalize(temp_mat, temp_mat, 0, 255, cv::NORM_MINMAX);
        temp_mat.convertTo(temp_mat, CV_8UC1);
        cv::applyColorMap(temp_mat, temp_mat, cv::COLORMAP_JET);

        scores = get_closest_matches_from_dist_mat(d_dist_mat_PM);

        return temp_mat;

    }

    // gets perfect memory distance matrix
    cv::Mat get_best_PM(cv::Mat &image) {
        cv::Mat img,current_image;
        cv::cvtColor(image, current_image, cv::COLOR_BGR2GRAY);
        cv::resize(current_image, current_image, {pm_width, pm_height},2);
        current_image.convertTo(img, CV_32FC1,(1.0)/255.0);
        int rows = pm_height;//img.rows;
        int cols = pm_width;//img.cols;
        int num_rotations = pm_width;
        get_rotations(img,num_rotations); // get all rotations of current image
        for (int i = 0; i < N_images;i++) {
            kernel_SAD_rotations<<<num_blocks,BLOCKSIZE>>>
                                         (d_images+i*rows*cols,
                                         d_rolled_images,
                                         d_temp,
                                         num_rotations,
                                         cols,
                                         rows);
            hipDeviceSynchronize();
            for (int j = 0; j < num_rotations; j++) {
                kernel_reduce<<<num_blocks,BLOCKSIZE>>>(d_temp+j*rows*cols, d_reduced_blocks+j*num_blocks, rows*cols);
            }

            hipDeviceSynchronize();
            kernel_reduce<<<num_rotations,num_blocks>>>(d_reduced_blocks, d_dist_mat_PM+i*num_rotations, num_blocks*num_rotations);
            hipDeviceSynchronize();

        }

        kernel_roll_image<<<N_images, num_rotations, num_rotations*sizeof(float) >>>(d_dist_mat_PM, d_dist_mat_PM, num_rotations,num_rotations/2);
        float temp_array[N_images*num_rotations];
        hipMemcpy(temp_array, d_dist_mat_PM, N_images*num_rotations*sizeof(float), hipMemcpyDeviceToHost);
        cv::Mat temp_mat(cv::Size(num_rotations, N_images), CV_32FC1, temp_array);
        cv::normalize(temp_mat, temp_mat, 0, 255, cv::NORM_MINMAX);
        temp_mat.convertTo(temp_mat, CV_8UC1);
        cv::applyColorMap(temp_mat, temp_mat, cv::COLORMAP_JET);
        return temp_mat;
    }

    // upload a hash sequence to the gpu*
    void uploadSequence(unsigned long long int *sequence) {
        l_sequence = sequence;
        gpuErrchk( hipMemcpy(d_sequence, l_sequence, d_sequence_size*sizeof(unsigned long long int), hipMemcpyHostToDevice));
    }

    // append one element to the hash sequence and pop front
    void addToSequence(unsigned long long int *hash) {

        kernel_shift_elements<<<(d_sequence_size+255/256),d_sequence_size>>>(d_sequence,d_tmp_seq, d_sequence_size);
        hipMemcpy(&d_sequence[d_sequence_size-1], hash, sizeof(unsigned long long int), hipMemcpyHostToDevice);
    }

    // get distance matrix between two datasets
    void getDistanceMatrix(unsigned long long int *sequence) {
        uploadSequence(sequence);
        kernel_construct_distance_matrix<<<(d_sequence_size+255/256), d_sequence_size>>>(d_sequence, d_hash_mat, d_cost_matrix, d_sequence_size, num_rows);
    }

    // gets the current distance matrix
    void getDistanceMatrix() {
        kernel_construct_distance_matrix<<<(d_sequence_size+255/256),d_sequence_size>>>(d_sequence, d_hash_mat, d_cost_matrix, d_sequence_size, num_rows);
        hipDeviceSynchronize();
    }



    // calculates accumulated cost matrix from a sequence of images and a pre-uploaded training hashes database
    static cv::Mat calculate_accumulated_cost_matrix(std::vector<cv::Mat> image_sequence, unsigned long long int *d_training_h, int num_training ) {
        int num_sequence = image_sequence.size();
        unsigned long long int *d_image_sequence;
        int *d_cost_matrix;
        int *d_cost_matrix_zigzag;
        int *d_D;
        int *d_D_ord;
        int *d_index;
        int *d_index_ord;
        hipMalloc(&d_image_sequence, num_sequence*sizeof(unsigned long long int));
        hipMalloc(&d_cost_matrix, num_sequence*num_training*sizeof(int));
        hipMalloc(&d_cost_matrix_zigzag, num_sequence*num_training*sizeof(int));
        hipMalloc(&d_index, num_sequence*num_training*sizeof(int));
        hipMalloc(&d_index_ord, num_sequence*num_training*sizeof(int));
        hipMalloc(&d_D, num_sequence*num_training*sizeof(int));
        hipMalloc(&d_D_ord, num_sequence*num_training*sizeof(int));

        // initialize GPU sequence matcher
        kernel_fill_index_matrix<<<num_training, num_sequence>>>(d_index_ord);
        hipDeviceSynchronize();
        kernel_order_dist_matrix<<<num_training, num_sequence, BLOCKSIZE*2>>>(d_index_ord, d_index, num_sequence, num_training);
        hipDeviceSynchronize();

        // calculate hashes and upload to gpu
        GpuDct gct(256);
        for (int i = 0; i < image_sequence.size(); i++) {
            auto hash = gct.dct(image_sequence[i]);
        }

        const int threads = 256;
        const int blocks = int(num_training+threads-1/threads);
        kernel_simple_dist_mat<<<blocks,threads>>>(d_training_h, d_image_sequence, d_cost_matrix, num_training, num_sequence);
        hipDeviceSynchronize();
        kernel_order_dist_matrix<<<num_training, num_sequence, BLOCKSIZE*2>>>(d_cost_matrix, d_cost_matrix_zigzag, num_sequence, num_training);
        hipDeviceSynchronize();
        kernel_calculate_accumulated_cost_matrix<<<1, num_sequence, 3*num_sequence>>>(d_D, d_cost_matrix_zigzag, num_training,num_sequence);
        hipDeviceSynchronize();
        // reordering the matrix (so it's human readable)
        kernel_reorder_matrix<<<num_training, num_sequence>>>(d_D_ord,d_D, d_index);

        //cv::cuda::GpuMat gpu_mat({num_sequence, num_training, CV_32SC1, d_D_ord});
        //cv::Mat host_mat;
       // gpu_mat.download(host_mat);
        int h_D[num_sequence][num_training];
        hipMemcpy(h_D, d_D_ord, num_sequence*num_training*sizeof(int),hipMemcpyDeviceToHost);
        cv::Mat host_mat(num_sequence, num_training, CV_8UC1, h_D);
        cv::normalize(host_mat, host_mat, 0, 255, cv::NORM_MINMAX);
        cv::applyColorMap(host_mat, host_mat, cv::COLORMAP_JET);
        cv::imshow("D", host_mat);
        cv::waitKey(0);
        return host_mat;

        hipFree(d_image_sequence);
        hipFree(d_cost_matrix);
        hipFree(d_cost_matrix_zigzag);
        hipFree(d_index);
        hipFree(d_index_ord);
        hipFree(d_D);
        hipFree(d_D_ord);
    }



    void calculate_accumulated_cost_matrix() {
        kernel_order_dist_matrix<<<num_rows, d_sequence_size, BLOCKSIZE*2>>>(d_cost_matrix, d_ordered_cost_mat, d_sequence_size, num_rows);
        hipDeviceSynchronize();
        kernel_calculate_accumulated_cost_matrix<<<1, d_sequence_size, 3*d_sequence_size>>>(d_accumulated_cost_mat, d_ordered_cost_mat, d_sequence_size,num_rows);
        hipDeviceSynchronize();
        // reordering the matrix (so it's human readable)
        kernel_reorder_matrix<<<num_rows, d_sequence_size>>>(d_accumulated_cost_mat_ord,d_accumulated_cost_mat, d_index_matrix);
    }



    // calculates the place and the rotation of a hash from a hash matrix
    std::pair<int,int> getMinIndex(std::bitset<64> current_hash, std::vector<std::bitset<64>> hashmat) {
        thrust::device_ptr<int> g_ptr =  thrust::device_pointer_cast(&d_accumulated_cost_mat_ord[(d_sequence_size-1)*num_rows]);
        int result_offset = thrust::min_element( g_ptr, g_ptr + (num_rows-1) ) -g_ptr;
        int min_value = *(g_ptr + result_offset);
        int min = 10000;
        int best_rot = -1;
        for (int i = 0; i < num_cols; i++) {
            if (result_offset >= 0 && result_offset < num_rows) {
                std::bitset<64> element = hashmat[result_offset*num_cols+i];

                int rot_hash_val = BoBRobotics::ImgProc::DCTHash::distance(element, current_hash);
                if (rot_hash_val < min) {
                    min = rot_hash_val;
                    best_rot = i;
                }
            }
        }
        std::cout << " ind " << result_offset << " rotation = " << best_rot << std::endl;
        return {result_offset, best_rot};
    }

    // downloads the accumulated cost matrix in a cv::Mat object
    cv::Mat downloadAccumulatedCostMatrix() {

        gpuErrchk( hipMemcpy(l_accumulated_cost_matrix, d_accumulated_cost_mat_ord, num_rows*d_sequence_size*sizeof(int), hipMemcpyDeviceToHost) );
        cv::cuda::GpuMat gpu_mat({d_sequence_size, num_rows, CV_32SC1, d_accumulated_cost_mat_ord});
        cv::Mat host_mat;
        gpu_mat.download(host_mat);

        return host_mat;
    }

    // downloads the currant distance matrix in opencv mat
    cv::Mat downloadDistanceMatrix() {
        gpuErrchk( hipMemcpy(l_cost_matrix, d_cost_matrix, num_rows*d_sequence_size*sizeof(int), hipMemcpyDeviceToHost) );
        cv::cuda::GpuMat gpu_mat({d_sequence_size, num_rows, CV_32SC1, d_cost_matrix});
        cv::Mat host_mat;
        gpu_mat.download(host_mat);
        return host_mat;
    }

    ~GPUHasher() {

        for (int i = 0; i < ull_pointers.size(); i++) {
            hipFree(ull_pointers[i]);
        }
        for (int i = 0; i < i_pointers.size(); i++) {
            hipFree(i_pointers[i]);
        }
        for (int i = 0; i < f_pointers.size(); i++) {
            hipFree(f_pointers[i]);
        }
    }

    private:

    std::vector<float *> f_pointers;
    std::vector<unsigned long long int*> ull_pointers;
    std::vector<int*> i_pointers;

    unsigned long long int *l_hash_mat; // host hash mat
    unsigned long long int *d_hash_mat; // device hash mat
    unsigned long long int *l_sequence; // sequence on host
    unsigned long long int *d_sequence; // sequence on device
    unsigned long long int *d_tmp_seq;  // temporary sequence holder
    int *d_ordered_cost_mat;            // dist matrix ordered in zig zag for parallel sequence matching
    int *l_cost_matrix;                 // final cost matrix on host
    int *l_accumulated_cost_matrix;
    int *d_cost_matrix;                 // final cost matrix on device
    int *d_index_matrix;                // hold the indices for the ordered cost matrix
    int *d_index_matrix_ord;
    int *d_accumulated_cost_mat;        // accumulated cost matrix
    int *d_accumulated_cost_mat_ord;    // accumulated cost matrix

    unsigned long long int *current_rotations;
    unsigned long long int *training_route;
    int *rot_dist_mat;

    float *d_T;
    unsigned long long int *l_best_row;
    float* d_rolled_images;  // all rotations of an image
    std::vector<uchar*> image_data_vector;
    hipblasHandle_t handle;

    // perfect memory-----
    float *d_temp;
    int num_blocks;
    float *d_reduced_blocks;
    float *d_SSD_rotations;
    float *d_dist_mat_PM;
    float *d_images; // image database
    int pm_width;
    int pm_height;
    int N_images;
    //----------------------

    //--- for single matches --
    float *d_training_images;
    float *d_testing_images;
    unsigned long long int *d_training_hashes;
    unsigned long long int *d_testing_hashes;
    int *d_single_distance_matrix;
    int N_training;
    int N_testing;
    int img_rows; // height of image
    int img_cols; // width of image

    unsigned int d_sequence_size;       // size of the sequence
    int N;                              // total size of a hash matrix with rotations
    int num_rows;                       // number of unique elements in a hash matrix
    int num_cols;                       // number of rotations in the training matrix
    uchar *d_image;
    uchar *d_rolled_image;
    uchar *l_rot_img_data;
    int m_image_width;
    int m_image_height;

};

